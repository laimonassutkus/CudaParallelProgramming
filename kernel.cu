
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string>
#include <type_traits>
#include <cassert>

#define LENGTH 10

template<class T> class DataContainer
{
public:
	DataContainer(T *_data, unsigned size);
	DataContainer(unsigned size);
	~DataContainer();

	T *getData() { return dataPtr; }
	unsigned getSize() { return dataSize; }

private:
	T *dataPtr;
	unsigned dataSize;
};

template<class T> DataContainer<T>::DataContainer(T *_data, unsigned size)
{
	dataPtr = _data;
	dataSize = size;
}

template<class T> DataContainer<T>::DataContainer(unsigned size)
{
	dataPtr = (T *)malloc(sizeof(T) * size);
	dataSize = size;
}

template<class T> DataContainer<T>::~DataContainer()
{
	free(dataPtr);
	dataPtr = nullptr;
}

template<typename T> hipError_t addWithCuda(DataContainer<T> &data, DataContainer<T> &results);
template<typename T> T cudaMallocWrapper(T devPtr, unsigned count);
template<typename T> bool cudaMemcpyWrapper(T target, T source, unsigned count, hipMemcpyKind cpyKind);
template<typename T> __global__ void addKernel(T dev_T, T result, unsigned size);

int *initSomeArray(unsigned size)
{
	int *a = (int *)malloc(sizeof(int) * size);
	std::fill(a, a + size, 1);
	return a;
}

int main()
{
	DataContainer<int> dataContainer(initSomeArray(LENGTH), LENGTH);
	DataContainer<int> resultContainer(LENGTH);

	hipError_t cudaStatus = addWithCuda(dataContainer, resultContainer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	puts("Results array:\n\n");
	for (unsigned i = 0; i < resultContainer.getSize(); i++)
	{
		printf("%d\n", resultContainer.getData()[i]);
	}

	return 0;
}

template<typename T> hipError_t addWithCuda(DataContainer<T> &data, DataContainer<T> &results)
{
	unsigned size = data.getSize();
	T *dev_dataPtr = nullptr;
	T *dev_resultPtr = nullptr;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed.");
		goto Error;
	}

	if ((dev_dataPtr = cudaMallocWrapper(dev_dataPtr, size)) == nullptr)
		goto Error;

	if ((dev_resultPtr = cudaMallocWrapper(dev_resultPtr, size)) == nullptr)
		goto Error;

	if (!cudaMemcpyWrapper(dev_dataPtr, data.getData(), size, hipMemcpyHostToDevice))
		goto Error;

	addKernel <<<1, size >>> (dev_dataPtr, dev_resultPtr, size);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	if (!cudaMemcpyWrapper(results.getData(), dev_resultPtr, size, hipMemcpyDeviceToHost))
		goto Error;

Error:
	hipFree(dev_dataPtr);
	hipFree(dev_resultPtr);

	return cudaStatus;
}

template<typename T> T cudaMallocWrapper(T devPtr, unsigned count)
{
	hipError_t cudaStatus = hipMalloc((void**)&devPtr, count * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return nullptr;
	}
	return devPtr;
}

template<typename T> bool cudaMemcpyWrapper(T target, T source, unsigned count, hipMemcpyKind cpyKind)
{
	hipError_t cudaStatus = hipMemcpy(target, source, count * sizeof(T), cpyKind);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}
	return true;
}

template<typename T> __global__ void addKernel(T dev_T, T result, unsigned size)
{
	int threadIndex = threadIdx.x;

	for (unsigned i = threadIndex; i < size; i++)
	{
		result[threadIndex] += dev_T[i];
	}
}

